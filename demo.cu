#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "math/math.cuh"
#include "math/primitives/activation_functions.cuh"
#include "math/primitives/loss_functions.cuh"

#include "network/network.cuh"

#include "utils/utils.cuh"

int main() {
  // Initialize network
  network_t *network = (network_t *) malloc(sizeof(network_t));
  size_t num_layers = 4;
  size_t num_inputs = 8; // 8 // 1024 // 100000;
  size_t num_outputs = 2;
  size_t num_epochs = 10;
  float lrate = 0.0004f;
  loss_func_t loss_func = CROSS_ENTROPY_LOSS;
  initialize_network(network, num_layers, num_inputs, num_outputs, num_epochs, lrate, loss_func);

  // Create layers
  create_append_layer(network, num_inputs, RELU);
  create_append_layer(network, 16, RELU); // 16 // 512 // 2048
  create_append_layer(network, 4, RELU); // 4 // 128 // 10240
  create_append_layer(network, num_outputs, SIGMOID);

  // Validate architecture
  if (!validate_network_arch(network)) {
    exit(1);
  }

  // Print network
  // print_network(network, false);

  // Feed network data
  float *data = (float *) malloc(sizeof(float) * num_inputs);
  fill_random_floats(data, num_inputs);
  feed_input_data(network, data);

  // Forward propagate
  // CPU
  clock_t start_cpu = clock();
  forward_propagate(network, true);
  clock_t end_cpu = clock();
  printf("CPU: %.3f\n", (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC);

  // GPU
  clock_t start_gpu = clock();
  forward_propagate(network, false);
  clock_t end_gpu = clock();
  printf("GPU: %.3f\n", (double)(end_gpu - start_gpu) / CLOCKS_PER_SEC);

  // Print network
  print_network(network, true);
}