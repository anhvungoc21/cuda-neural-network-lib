#include "errors.cuh"

/**
 * Assert-style function to handle errors from CUDA API calls
 * Source:
 * https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
 */
void gpuAssert(hipError_t code, const char *file, int line, bool abort) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}