#include "hip/hip_runtime.h"
#include "utils.cuh"

/**
 * Fills an array with random whole-number floats from 1 to 10
 */
void fill_random_ints(float *arr, int size) {
  for (int i = 0; i < size; i++) {
    arr[i] = (float)(rand() % 10 + 1);
  }
}

/**
 * Fills an array with random floats from 1 to 10
 */
void fill_random_floats(float *arr, int size) {
  for (int i = 0; i < size; i++) {
    arr[i] = 1.0f + ((float)rand() / RAND_MAX) * 9.0f;
  }
}

/**
 * Pretty prints a matrix
 */
void print_matrix(float *mat, int rows, int cols) {
  for (int r = 0; r < rows; r++) {
    for (int c = 0; c < cols; c++) {
      printf("%.3f ", mat[r * cols + c]);
    }
    printf("\n");
  }
  printf("\n");
}

/**
 * Prints an array
 */
void print_arr(float *arr, int size) {
  for (int i = 0; i < size; i++) {
    printf("%.1f ", arr[i]);
  }
  printf("\n\n");
}

/**
 * Checks whether 2 floats are almost equal
 */
int almostEquals(float a, float b, float epsilon) {
  return fabs(a - b) < epsilon;
}

/**
 * Checks 2 arrays for equality
 */
void check_equal_arr(float *arr1, float *arr2, int size, float epsilon) {
  for (int i = 0; i < size; i++) {
    if (!almostEquals(arr1[i], arr2[i], epsilon)) {
      printf("Unequal: %.6f vs. %.6f\n", arr1[i], arr2[i]);
      return;
    }
  }

  printf("Equal!\n");
}