#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "math/gpu_math.cuh"

extern "C" {
#include "math/cpu_math.h"
#include "utils/utils.h"
}


int main() {
  srand(time(NULL));

  int rows_A = 1024;
  int cols_A = 512;
  int rows_B = 512;
  int cols_B = 2048;

  float *A = (float *) malloc(sizeof(float) * (rows_A * cols_A));
  float *B = (float *) malloc(sizeof(float) * (rows_B * cols_B));
  float *cpu_result = (float *) malloc(sizeof(float) * (rows_A * cols_B));
  float *gpu_result = (float *) malloc(sizeof(float) * (rows_A * cols_B));

  fill_random_floats(A, rows_A * cols_A);
  fill_random_floats(B, rows_B * cols_B);

  // CPU
  clock_t start_cpu = clock();
  cpu__matrix_multiply(A, B, cpu_result, rows_A, cols_A, rows_B, cols_B);
  clock_t end_cpu = clock();
  printf("CPU: %.3f\n", (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC);

  // GPU
  clock_t start_gpu = clock();
  gpu__matrix_multiply(A, B, gpu_result, rows_A, cols_A, rows_B, cols_B);
  clock_t end_gpu = clock();
  printf("GPU: %.3f\n", (double)(end_gpu - start_gpu) / CLOCKS_PER_SEC);

  // Check for result equality
  float tolerance = 1e-6;
  check_equal_arr(cpu_result, gpu_result, rows_A * cols_B, tolerance);

  free(A);
  free(B);
  free(cpu_result);
  free(gpu_result);
}
