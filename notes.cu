
#include <hip/hip_runtime.h>
// SIMT: Simple Instruction, Multiple Threads

// Architecture
// Threads
//  - Lowest granualarity
//  - Execute instructions
// Warps
//  - Lowest SCHEDULABLE granularity
//  - Executes same instructions together (lock-step)
// Thread blocks
//  - Lowest PROGRAMMABLE entity
//  - Assigned to a single shader core
// Grids
//  - How a problem is mapped to the GPU
//  - Part of GPU LAUNCH PARAMETERS (#Blocks, #Threads

// Matrix Multiplication
// -> Think in terms of the resulting matrix!
//    Each thread is responsible for one cell of that matrix
//    -> One row of first matrix, One column of second matrix
//    -> Only ever need ONE for-loop for each thread

// Cache Tiling
// -> Shared Memory (Scratchpad)
//    - User-managed L1 Cache
//    - Private per block
// => Basically we copy each "tile" of the matrix to its corresponding block's
// shared memory
// => This way we always access cache instead of memory

// Coalescing
// - In terms of memory addresses, matrices are in row-major order cuz it's 1D
// of rows
// => Matrix A: Each thread accesses a different ROW => misaligned
// => Matrix B: Each thread accesses a different COLUMN => aligned (adjacent)
//              Multiple adjacent accesses can be coalesced into a single wide
//              access
// ==> Solution: Transpose the A Matrix!!.
// This doesn't help crazily though

// L1, L2 caches?


// Single vs Double precision?
// https://www.geeksforgeeks.org/difference-between-single-precision-and-double-precision/


// INSANE GUIDE:
// https://siboehm.com/articles/22/CUDA-MMM