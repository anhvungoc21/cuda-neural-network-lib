#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../math/gpu_math.cuh"

extern "C" {
#include "../math/cpu_math.h"
#include "../utils/utils.h"
}

// === Findings ===
// Using Grinnell MathLAN machines, the GPU starts to perform similarly to the
// CPU when the resulting matrix of a matrix multiplication reaches around 8192
// cells. Before this point, it is not worth using the GPU due to the overhead
// of copying memory back and forth.

int main() {
  srand(time(NULL));

  int rows_A = 2048;
  int cols_A = 4;
  int rows_B = 4;
  int cols_B = 2048;

  float *A = (float *)malloc(sizeof(float) * (rows_A * cols_A));
  float *B = (float *)malloc(sizeof(float) * (rows_B * cols_B));
  float *cpu_result = (float *)malloc(sizeof(float) * (rows_A * cols_B));
  float *gpu_result = (float *)malloc(sizeof(float) * (rows_A * cols_B));

  fill_random_floats(A, rows_A * cols_A);
  fill_random_floats(B, rows_B * cols_B);

  // CPU
  clock_t start_cpu = clock();
  cpu__matrix_multiply(A, B, cpu_result, rows_A, rows_B, cols_B);
  clock_t end_cpu = clock();
  printf("CPU: %.3f\n", (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC);

  // GPU
  clock_t start_gpu = clock();
  gpu__matrix_multiply(A, B, gpu_result, rows_A, rows_B, cols_B);
  clock_t end_gpu = clock();
  printf("GPU: %.3f\n", (double)(end_gpu - start_gpu) / CLOCKS_PER_SEC);

  free(A);
  free(B);
  free(cpu_result);
  free(gpu_result);
}
