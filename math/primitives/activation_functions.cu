#include "hip/hip_runtime.h"
#include "activation_functions.cuh"

#include <math.h>
#include <stdio.h>

/**
 * Sigmoid activation function
 */
float sigmoid(float x) {
  // Handle too big/small input
  if (x >= 45.0f)
    return 1.0f;
  if (x <= -45.0f)
    return 0.0f;

  return 1.0f / (1.0f + exp(-x));
}

/**
 * Rectified Linear Unit activation function
 */
float reLU(float x) { return fmaxf(0.0f, x); }

/**
 * Softmax activation function
 */
void softMax(float *arr, size_t size) {
  float sum_exp = 0.0f;
  
  // Calculate and sum exponents
  for (int i = 0; i < size; i++) {
    arr[i] = exp(arr[i]);
    sum_exp += arr[i];
  }

  // Divide exponents by sum
  for (int i = 0; i < size; i++) {
    arr[i] /= sum_exp;
  }
}

/** 
 * Derivative of Sigmoid activation function
 */
float sigmoid_derivative(float output) {
  return output * (1.0f - output);
}

/** Derivative of ReLU activation function
 *
 */
float reLU_derivative(float output) {
  return (output > 0.0f) ? 1.0f : 0.0f;
}

/**
 * Translate activation_func_t enum to string representation
 */
void activation_func_to_string(activation_func_t acti_func) {
  if (acti_func == RELU) {
    printf("ReLU\n");
  } else if (acti_func == SIGMOID) {
    printf("Sigmoid\n");
  }
} 