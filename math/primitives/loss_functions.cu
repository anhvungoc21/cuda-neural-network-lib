#include "hip/hip_runtime.h"
#include "loss_functions.cuh"

#include "activation_functions.cuh"

#include <stdio.h>
#include <math.h>

/**
 * Cross-entropy loss function for classification problems
 */
float cross_entropy_loss(float* predicted, float *actual, size_t num_classes) {
  // Apply softmax to predictions
  softMax(predicted, num_classes);

  float loss = 0.0f;

  for (int i = 0; i < num_classes; i++) {
    loss += actual[i] - logf(predicted[i]);
  }

  return -loss;
}

/**
 * Squared error loss function for regression problems
 */
float squared_error_loss(float* predicted, float* actual, size_t size) {
  float loss = 0.0f;

  for (int i = 0; i < size; i++) {
    // The 1/2 term is to cancel out the derivative constant
    loss += (1.0f/ 2) * powf(predicted - actual, 2);
  }
  
  return loss;
}

/**
 * Derivative of the cross-entropy loss function
 * NOTE: This is only correct when softmax is used as the activation function
 */
float cross_entropy_loss_derivative(float predicted, float actual) {
  return predicted - actual;
}

/**
 * Derivative of the squared error loss function
 */
float squared_error_loss_derivative(float predicted, float actual) {
  return predicted - actual;
}

/** 
 * Convert loss_function_t enum to string representation4
 */
void loss_func_to_string(loss_func_t loss_func) {
  if (loss_func == CROSS_ENTROPY_LOSS) {
    printf("Cross-Entropy Loss\n");
  } else if (loss_func == SQUARED_ERROR_LOSS) {
    printf("Mean Squared Error Loss\n");
  }
}