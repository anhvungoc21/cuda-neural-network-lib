#include "hip/hip_runtime.h"
#include "../utils/errors.cuh"

#include "gpu_math.cuh"

/**
 * Kernel for matrix multiplication
 *
 * @param A First matrix (m x p)
 * @param B Second matrix (p x n)
 * @param result Result matrix (m x n)
 */
__global__ void __kernel_matrix_multiply(float *A, float *B, float *result,
                                         size_t rows_A, size_t inner_dim,
                                         size_t cols_B) {
  // Get row and column of thread in result matrix
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;

  // Check boundaries
  if (row >= rows_A || col >= cols_B)
    return;

  // Calculate cell
  float sum = 0.0f;
  for (int k = 0; k < inner_dim; k++) {
    sum += A[row * inner_dim + k] * B[k * cols_B + col];
  }

  // Store result
  result[row * cols_B + col] = sum;
}

/**
 * Performs a matrix multiplication on the GPU
 *
 * @param A First matrix (m x p)
 * @param B Second matrix (p x n)
 * @param result Result matrix (m x n)
 */
void gpu__matrix_multiply(float *A, float *B, float *result, size_t rows_A,
                          size_t inner_dim, size_t cols_B) {
  // Set up on GPU
  float *gpu_A, *gpu_B, *gpu_result;
  size_t size_A = sizeof(float) * (rows_A * inner_dim);
  size_t size_B = sizeof(float) * (inner_dim * cols_B);
  size_t size_result = sizeof(float) * (rows_A * cols_B);

  gpuErrchk(hipMalloc(&gpu_A, size_A));
  gpuErrchk(hipMalloc(&gpu_B, size_B));
  gpuErrchk(hipMalloc(&gpu_result, size_result));

  gpuErrchk(hipMemcpy(gpu_A, A, size_A, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(gpu_B, B, size_B, hipMemcpyHostToDevice));
  gpuErrchk(
      hipMemcpy(gpu_result, result, size_result, hipMemcpyHostToDevice));

  // Each block has a fixed 32 x 32 threads
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

  // Each grid has a X and Y corresponding to the shape of result matrix
  // The plus one extra unit and - 1 is to ensure rounding up
  size_t gridCols = (cols_B + blockSize.x - 1) / blockSize.x;
  size_t gridRows = (rows_A + blockSize.y - 1) / blockSize.y;
  dim3 gridSize(gridCols, gridRows);

  // Run kernel
  __kernel_matrix_multiply<<<gridSize, blockSize>>>(gpu_A, gpu_B, gpu_result,
                                                    rows_A, inner_dim, cols_B);

  // Copy result back to CPU
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(
      hipMemcpy(result, gpu_result, size_result, hipMemcpyDeviceToHost));

  // Clean up
  hipFree(gpu_A);
  hipFree(gpu_B);
  hipFree(gpu_result);
}
