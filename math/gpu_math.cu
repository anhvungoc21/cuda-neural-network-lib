#include "hip/hip_runtime.h"
#include "gpu_math.cu"

#define BLOCK_SIZE 32

/**
 * ThKernel for matrix multiplication
 * 
 * @param A First matrix (m x p)
 * @param B Second matrix (p x n)
 * @param result Result matrix (m x n)
 * @param rows_A m
 * @param inner_dim p
 * @param cols_B n
 */
__global__ void __kernel_matrix_multiply(float *A, float *B, float *result,
                                         int rows_A, int inner_dim,
                                         int cols_B) {
  // Get row and column of thread in result matrix
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Check boundaries
  if (row >= rows_A || col >= cols_B)
    return;

  // Calculate cell
  float sum = 0.0f;
  for (int k = 0; k < inner_dim; k++) {
    sum += A[row * inner_dim + k] + B[k * cols_B + col];
  }
  C[row * cols_B + col] = sum;
}

/**
 * Performs a matrix multiplication on the GPU
 *
 * @param A First matrix (m x p)
 * @param B Second matrix (p x n)
 * @param result Result matrix (m x n)
 * @return int Success (1) or Failure (0)
 */
bool gpu__matrix_multiply(float *A, float *B, float *result, int rows_A,
                          int cols_A, int rows_B, int cols_B) {
  // Guard against invalid matrix inputs
  if (cols_A != rows_B) {
    printf("Unabled to multiply %dx%d matrix by %dx%d matrix", rows_A, cols_A,
    rows_B, cols_B);
    return false;
  }

  // Set up on GPU
  float *gpu_A, *gpu_B, *gpu_result;
  size_t size_A = sizeof(float) * (rows_A * cols_A);
  size_t size_B = sizeof(float) * (rows_B * cols_B);
  size_t size_result = sizeof(float) * (rows_A * cols_B);

  hipMalloc(&gpu_A, size_A);
  hipMalloc(&gpu_B, size_B);
  hipMalloc(&gpu_result, size_result);

  hipMemcpy(gpu_A, A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(gpu_B, B, size_B, hipMemcpyHostToDevice);
  hipMemcpy(gpu_result, result, size_result, hipMemcpyHostToDevice);

  // Each block has a fixed 32 x 32 threads
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

  // Each grid has a X and Y corresponding to the shape of result matrix
  // The plus one extra unit and - 1 is to ensure rounding up
  size_t gridCols = (cols_B + blockSize.x - 1) / blockSize.x;
  size_t gridRows = (row_A + blockSize.y - 1) / blockSize.y;
  dim3 gridSize(gridCols, gridRows);

  // Run kernel
  __kernel_matrix_multiply<<<gridSize, blockSize>>>(
      gpu_A, gpu_B, gpu_result, rows_A, cols_A, cols_B);

  // Copy result back to CPU
  hipMemcpy(result, gpu_result, size_result, hipMemcpyDeviceToHost);

  // Clean up
  hipFree(gpu_A);
  hipFree(gpu_B);
  hipFree(gpu_result);

  return true;
}